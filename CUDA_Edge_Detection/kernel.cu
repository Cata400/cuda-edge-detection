#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cstring>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>

using namespace cv;

__constant__ float gx[3][3] = { { 1, 0, -1 }, { 2, 0, -2 }, { 1, 0, -1 } };
__constant__ float gy[3][3] = { { 1, 2, 1 },  { 0, 0, 0 }, { -1, -2, -1 } };


__global__ void edge_detect_cuda(uchar* input, float* output, int height, int width, bool pixel) {
    int i, j;

    if (pixel) {
        i = blockIdx.x;
        j = threadIdx.x;
    }
    else {
        i = blockIdx.x * blockDim.x + threadIdx.x;
        j = blockIdx.y * blockDim.y + threadIdx.y;
    }

    float grad_x = 0, grad_y = 0;

    if (i > 0 && i < height - 1 && j > 0 && j < width - 1) {
        for (int k = -1; k < 2; k++) {
            for (int l = -1; l < 2; l++) {
				grad_x += input[(i + k) * width + (j + l)] * gx[k + 1][l + 1];
				grad_y += input[(i + k) * width + (j + l)] * gy[k + 1][l + 1];
			}
		}
		output[i * width + j] = sqrt(grad_x * grad_x + grad_y * grad_y);
    }
    else {
        output[i * width + j] = 0;
    }
}


int main()
{
    // Parameters
    std::string image_name = "Lenna_multiplied_4x4";
    std::string image_path = "../Images/" + image_name + ".png";
    std::string save_path_cuda = "../Images/" + image_name + "_edge_cuda.png";
    int reps = 100;
    bool pixel = false;

    // Read image
    Mat image = imread(image_path, IMREAD_COLOR);

    if (!image.data) {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }
    imshow("Image to detect edges from", image);
    waitKey();
    destroyAllWindows();

    cvtColor(image, image, COLOR_BGR2GRAY);

    // Allocate arrays and dimensions
    int height = image.rows;
    int width = image.cols;

    int block_size_x = 8;
    int block_size_y = 8;
    dim3 block_size(block_size_x, block_size_y);

    int grid_size_x = (height + block_size_x - 1) / block_size_x;
    int grid_size_y = (width + block_size_y - 1) / block_size_y;
    dim3 grid_size(grid_size_x, grid_size_y);

    uchar* image_data = new uchar[height * width];
    int index = 0;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            image_data[index++] = image.at<uchar>(i, j);
        }
    }

    float* image_edges = new float[height * width];
    Mat edges_cuda;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Allocate GPU buffers for input and output image
    uchar* image_data_cuda;
    float* image_edges_cuda;

    cudaStatus = hipMalloc((void**)&image_data_cuda, height * width * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&image_edges_cuda, height * width * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(image_data_cuda, image_data, height * width * sizeof(uchar), hipMemcpyHostToDevice); // <---- error here
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    // Launch a kernel on the GPU with one thread for each pixel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < reps; i++) {
        if (pixel) {
            edge_detect_cuda <<< height, width >>> (image_data_cuda, image_edges_cuda, height, width, pixel);
        }
        else {
            edge_detect_cuda <<< grid_size, block_size >>> (image_data_cuda, image_edges_cuda, height, width, pixel);
        }

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
            goto Error;
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "CUDA edge detection took " << elapsedTime << " ms" << std::endl;
    
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(image_edges, image_edges_cuda, height * width * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    // Save image
    edges_cuda = Mat(height, width, CV_32FC1, image_edges);
    Rect roi(1, 1, width - 2, height - 2);
    edges_cuda = edges_cuda(roi);
    imwrite(save_path_cuda, edges_cuda);

    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    delete[] image_edges;
    delete[] image_data;
    

Error:
    hipFree(image_data_cuda);
    hipFree(image_edges_cuda);

    return cudaStatus;
}

